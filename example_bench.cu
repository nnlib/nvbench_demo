#include "hip/hip_runtime.h"
#include <nvbench/nvbench.cuh>

#include <cuda/std/chrono>

#include <hip/hip_runtime.h>

#include <nvbench/nvbench.cuh>

// Grab some testing kernels from NVBench:
#include <nvbench/test_kernels.cuh>

// Thrust vectors simplify memory management:
#include <thrust/device_vector.h>


__global__ void sleep_kernel(nvbench::int64_t microseconds) {
  const auto start = cuda::std::chrono::high_resolution_clock::now();
  const auto target_duration = cuda::std::chrono::microseconds(microseconds);
  const auto finish = start + target_duration;

  auto now = cuda::std::chrono::high_resolution_clock::now();
  while (now < finish) {
    now = cuda::std::chrono::high_resolution_clock::now();
  }
}

void sleep_benchmark(nvbench::state &state) {
  const auto duration_us = state.get_int64("Duration (us)");
  state.exec([&duration_us](nvbench::launch &launch) {
    sleep_kernel<<<1, 1, 0, launch.get_stream()>>>(duration_us);
  });
}

void throughput_bench(nvbench::state &state)
{
  // Allocate input data:
  const std::size_t num_values = 640 * 1024 * 1024 / sizeof(nvbench::int32_t);
  thrust::device_vector<nvbench::int32_t> input(num_values);
  thrust::device_vector<nvbench::int32_t> output(num_values);

  // Provide throughput information:
  state.add_element_count(num_values, "NumElements");
  state.add_global_memory_reads<nvbench::int32_t>(num_values, "DataSize");
  state.add_global_memory_writes<nvbench::int32_t>(num_values);

  state.exec([&input, &output, num_values](nvbench::launch &launch) {
    nvbench::copy_kernel<<<256, 256, 0, launch.get_stream()>>>(
      thrust::raw_pointer_cast(input.data()),
      thrust::raw_pointer_cast(output.data()),
      num_values);
  });
}

NVBENCH_BENCH(sleep_benchmark)
    .add_int64_axis("Duration (us)", nvbench::range(0, 100, 5))
    .set_timeout(1); // Limit to one second per measurement.

NVBENCH_BENCH(throughput_bench);
